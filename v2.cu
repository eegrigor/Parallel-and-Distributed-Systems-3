

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

int *unroll(int **ising, int n){
    int *ising1d = (int *)malloc(n * n * sizeof(int));
    for(int i = 0 ; i < n ; i++){
        for(int j = 0 ; j < n ; j++){
            ising1d[i*n + j] = ising[i][j];
        }
    }

    return ising1d;
}
void swap(int  **a, int  **b) {
  int  *tmp = *a;
  *a = *b;
  *b = tmp;
}

__global__ void moment(int *ising, int *newising, int n, int b){
      for(int i = ((blockIdx.x*1024 + threadIdx.x)*b/n)*b ; i < ((blockIdx.x*1024 + threadIdx.x)*b/n)*b + b; i++){
        for(int j = ((blockIdx.x*1024 + threadIdx.x)%(n/b))*b ; j < ((blockIdx.x*1024 + threadIdx.x)%(n/b))*b + b; j++){
          int sum = ising[i*n + j + n - n*n*(i==n-1)] + ising[i*n + j - n + n*n*(i==0)]
          + ising[i*n + j + 1 - n*(j%n == n - 1)]
          + ising[i*n + j - 1 + n*(j%n == 0)]
          + ising[i*n + j];
          if(sum > 0)
            newising[i*n + j] = 1 ;
          else
            newising[i*n + j] = -1 ;
        }
      }
}

int main(int argc, char **argv){

    //size of Ising model
    int n = 2048;
    // number of iterations
    int k = 100;

    srand(time(NULL));

    int *ising = (int *) malloc(n * n * sizeof(int));
    for(int i = 0 ; i < n ; i++){
        for(int j = 0 ; j < n ; j ++){
            ising[i*n + j] = rand() % 2 ;
            if(ising[i*n + j] == 0){
                ising[i*n + j] = -1;
            }
        }
    }

    /*for(int i = 0 ; i < n ; i++){
        for(int j = 0 ; j < n ; j++){
            printf("%d " , ising[i*n + j]);
        }
        printf("\n");
    }
    printf("\n");*/

    int *newising = (int *)malloc(n * n * sizeof(int));
    
    int *d_ising;
    int *d_newising;
    int size = n * n * sizeof(int);
    
    //allocate on gpu
    hipMalloc((void**)&d_ising, size);
    hipMalloc((void**)&d_newising, size);
    
    //b size
    int b = 32;

    int blocks = ((n*n/(b*b))-1)/1024 + 1;
    
    struct timeval start, end;
    double time;
    

    for(int l = 0 ; l < k ; l++){
        //copy data to gpu
        hipMemcpy(d_ising, ising, size, hipMemcpyHostToDevice);
        //call function on gpu with n*n threads
        gettimeofday(&start, NULL);
        moment<<<blocks,(n*n/(b*b))/blocks>>>(d_ising, d_newising, n, b);
        gettimeofday(&end, NULL);
        //copy result from gpu
        hipMemcpy(newising, d_newising, size, hipMemcpyDeviceToHost);
        time += (double)((end.tv_usec - start.tv_usec)/1.0e6 + end.tv_sec - start.tv_sec);

        swap(&ising,&newising);

       
    }

     /*for(int i = 0 ; i < n ; i++){
            for(int j = 0 ; j < n ; j++){
                printf("%d " , ising[i*n + j]);
            }
            printf("\n");
        }
        printf("\n");*/
        
    
    printf("time: %f\n", time);

    //free pointers
    free(ising);
    free(newising);
    hipFree(d_ising);
    hipFree(d_newising);
    return 0 ;
}
